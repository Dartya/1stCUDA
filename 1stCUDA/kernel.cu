
#include "hip/hip_runtime.h"


#include <stdio.h>

//��������������� ������� ��� ������������� CUDA ��� ������������� ������������ ��������.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

//�������, ������� ������ ��������� GPU
__global__ void addKernel(int *c, const int *a, const int *b)
{
	//� �������� ���������� ������� ������� ������������ ������ ���� � �����
    int i = threadIdx.x;	//�������� ������ ���� � �����
    c[i] = a[i] + b[i];		//���������� �������� ��� ���������������� ������� ���������� ��������
}

int main()
{
	//������������� ��������: ��������� ������ ���������� �������� �������� � � b 
    const int arraySize = 5;							//���������� ����� ��������
    const int a[arraySize] = { 1, 2, 3, 4, 5 };			//���� ������� �
    const int b[arraySize] = { 10, 20, 30, 40, 50 };	//� b
    int c[arraySize] = { 0 };							//� �������������� ������

    // �������� �������� � ������������ ������� (�� GPU) - ���������� ��������������� �������
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	//����� ���������� ����������
    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

	// 5.5 ����� ����������

    // cudaDeviceReset ������ ���� ������ ����� ������� �� ���������, ����� ����������� �������������� � �����������
	// ���������� ������ traces
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// ��������������� ������� ��� ������������� CUDA ��� ������������� ������������ ��������.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;
	// 0. ��������������� ����������

    // �������, � ����� GPU ��������. � ������-��� ������� ����� �������� �������� ������
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	//1. ���� �������� ����������� ���������� ������ �� ����������

    // ��������� ������� ��� ���� �������� (��� �������, ���� ��������)
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int)); //�������� size(� ����� ������ ������ �������� arraySize) ������ ������ �� ���������� (���) � ���������� ��������� �� ���������� ������ � dev_c
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	//�� �� ����� ��� ������� �
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	//�� �� ����� ��� ������� b
    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	//2. ���� �������� ������ �� ����� ������ � ������ ����������

    // ����������� �������� �������� � ������� �� ������ ����� � ������ ���
	//cudaMemcpy() �������� size * sizeof(int) ������ �� ������, �� ������� ��������� � � ������, �� ������� ��������� dev_a, ����������� � ����� �� ����������
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);	
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	//�� �� ����� ��� ������� b
    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	// 3. ���� ��������� ���� �� ���������� //4 ���������� ��������� ���� *����� �� �����, ������ ����� ��������� �� ������* 

    // ���� ��������� ���� �� ��� � ����� ������� ��� ������� ��������
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);	//����� ������������ ����� � ��������� __global__ ������� ����, ���
	//1 - ������ ����� � ������, size - ������ ������� ����� (� ����� ������ arraySize = 5)

    // �������� �� ������� ������ ��� ������� ����
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize ������� ���������� ������ ���� � ���������� ��� ������, ������������ �� ����� �������.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	// 5. ���� �������� ���������� �� ������ ���������� � ���� ������

    // ����������� ������� �� ������ GPU � ������ �����
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);	//�������� �������� - ����� ��� ����������� ����������� �� ���������� � ����
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	
Error: //� ������ ����������� ������ �� ������ �� ������ ���� ��� ���������� ��� �� ��� ����� (� ���������� �� �����������)
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
